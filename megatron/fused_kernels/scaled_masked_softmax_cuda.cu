/* coding=utf-8
 * Copyright (c) 2020, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include "THC/THC.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "scaled_masked_softmax.h"

namespace multihead_attn {
namespace fused_softmax {
namespace scaled_masked_softmax {

torch::Tensor fwd_cuda(
    torch::Tensor const& input,
    torch::Tensor const& mask,
    float scale_factor)
{
  // input is a 4d tensor with dimensions [batches, attn_heads, seq_len, seq_len]
  const int batches = input.size(0);
  const int pad_batches = mask.size(0);
  const int attn_heads = input.size(1);
  const int seq_len = input.size(2);
  TORCH_INTERNAL_ASSERT(seq_len <= 2048);
  TORCH_INTERNAL_ASSERT(pad_batches == 1 || pad_batches == batches);
  TORCH_INTERNAL_ASSERT(mask.size(1) == 1);
  TORCH_INTERNAL_ASSERT(mask.size(2) == seq_len);
  TORCH_INTERNAL_ASSERT(mask.size(3) == seq_len);

  // Output 
  auto act_options = input.options().requires_grad(false);
  torch::Tensor softmax_results = 
      torch::empty({batches, attn_heads, seq_len, seq_len}, act_options);

  // Softmax Intermediate Result Ptr
  void* input_ptr = static_cast<void*>(input.data_ptr());
  void* mask_ptr = static_cast<void*>(mask.data_ptr());
  void* softmax_results_ptr = static_cast<void*>(softmax_results.data_ptr());

  dispatch_scaled_masked_softmax_forward<half, half, float>(
      reinterpret_cast<half*>(softmax_results_ptr),
      reinterpret_cast<const half*>(input_ptr),
      reinterpret_cast<const uint8_t*>(mask_ptr),
      scale_factor,
      seq_len,
      seq_len,
      batches,
      attn_heads,
      pad_batches);
  return softmax_results;
}

torch::Tensor bwd_cuda(
    torch::Tensor const& output_grads_, 
    torch::Tensor const& softmax_results_, 
    float scale_factor)  {
	
  auto output_grads = output_grads_.contiguous();
  auto softmax_results = softmax_results_.contiguous();

  //output grads is a 4d tensor with dimensions [batches, attn_heads, seq_len, seq_len]
  const int batches = output_grads.size(0);
  const int attn_heads = output_grads.size(1);
  const int seq_len = output_grads.size(2);
  TORCH_INTERNAL_ASSERT(output_grads.size(2) == output_grads.size(3));

  void* output_grads_ptr = static_cast<void*>(output_grads.data_ptr());

  //Softmax Grad
  dispatch_scaled_masked_softmax_backward<half, half, float>(
      reinterpret_cast<half*>(output_grads_ptr), 
      reinterpret_cast<half*>(output_grads_ptr), 
      reinterpret_cast<half const*>(softmax_results.data_ptr()),
      scale_factor,
      seq_len,
      seq_len,
      batches,
      attn_heads);
  
  //backward pass is completely in-place
  return output_grads;
}
}
}
}
